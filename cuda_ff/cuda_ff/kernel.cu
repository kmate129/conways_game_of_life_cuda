#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include "GameSpace.h"
#include ""
#include "hip/hip_runtime.h"

const int _dimension = 10;
const int _threads = 4;
const int _goal = 1;

typedef int my_arr[_dimension];

__device__ int test;

__global__ void NextRound(my_arr* table, my_arr* temptable, int dimension) {
	int global_column = blockIdx.x * blockDim.x + threadIdx.x;
	int global_row = blockIdx.y * blockDim.y + threadIdx.y;

	if (global_row >= dimension || global_column >= dimension) {
		return;
	}

	__shared__ int shr_matrix[_threads + 2][_threads + 2];
	shr_matrix[threadIdx.y + 1][threadIdx.x + 1] = table[global_row][global_column];

	if (threadIdx.x == 0 && global_column != 0) 
	{
		shr_matrix[threadIdx.y + 1][threadIdx.x] = table[global_row][global_column-1];

		if (threadIdx.y == 0) 
		{
			shr_matrix[threadIdx.y][threadIdx.x] = table[global_row-1][global_column-1];
		}
		if (threadIdx.y == _threads - 1) {
			shr_matrix[threadIdx.y+2][threadIdx.x] = table[global_row+1][global_column-1];
		}
	}
	if (threadIdx.x == _threads - 1 && global_column != dimension - 1) 
	{
		shr_matrix[threadIdx.y+1][threadIdx.x+2] = table[global_row][global_column + 1];

		if (threadIdx.y == 0) 
		{
			shr_matrix[threadIdx.y][threadIdx.x+2] = table[global_row-1][global_column+1];
		}
		if (threadIdx.y == _threads - 1)
		{
			shr_matrix[threadIdx.y+2][threadIdx.x+2] = table[global_row+1][global_column+1];
		}
	}
	if (threadIdx.y == 0 && global_row != 0)
	{
		shr_matrix[threadIdx.y][threadIdx.x+1] = table[global_row-1][global_column];
	}
	if (threadIdx.y == _threads - 1 && global_row != dimension - 1)
	{
		shr_matrix[threadIdx.y+2][threadIdx.x+1] = table[global_row+1][global_column];
	}

	int count = 0;
	int cell = shr_matrix[threadIdx.y + 1][threadIdx.x + 1];

	int startRow = threadIdx.y;
	int startColumn = threadIdx.x;
	int endRow = threadIdx.y + 2;
	int endColumn = threadIdx.x + 2;

	if (global_row == 0)
	{
		startRow = threadIdx.y + 1;
	}
	if (global_row == dimension - 1)
	{
		endRow = threadIdx.y + 1;
	}
	if (global_column == 0)
	{
		startColumn = threadIdx.x + 1;
	}
	if (global_column == dimension - 1)
	{
		endColumn = threadIdx.x + 1;
	}

	for (int k = startRow; k <= endRow; ++k)
	{
		for (int l = startColumn; l <= endColumn; ++l)
		{
			if ((k != threadIdx.y + 1 || l != threadIdx.x + 1) && shr_matrix[k][l] == 1)
			{
				count++;
			}
		}
	}

	if (cell == 1 && (count == 2 || count == 3))
	{
		temptable[global_row][global_column] = 1;
	}
	else if (cell == 1 && (count < 2 || count >3))
	{
		temptable[global_row][global_column] = 0;
	}
	else if (cell == 0 && count == 3)
	{
		temptable[global_row][global_column] = 1;
	}
	else
	{
		temptable[global_row][global_column] = 0;
	}
}

__global__ void kernel(my_arr* table, my_arr* temptable, int dimension) {
	int global_column = blockIdx.x * blockDim.x + threadIdx.x;
	int global_row = blockIdx.y * blockDim.y + threadIdx.y;

	if (global_row >= dimension || global_column >= dimension) {
		return;
	}

	table[global_row][global_column] = temptable[global_row][global_column];

}

int main()
{
	GameSpace g(_dimension, _dimension, 0.3);

	my_arr* host_table;
	my_arr* dev_table;
	my_arr* dev_temptable;
	size_t dsize = _dimension * _dimension * sizeof(int);
	host_table = (my_arr*)malloc(dsize);

	for (size_t i = 0; i < _dimension; ++i) {
		for (size_t j = 0; j < _dimension; ++j) {
			host_table[i][j] = g.GetTable()[i][j];
		}
	}

	for (size_t i = 0; i < _dimension; ++i) {
		for (size_t j = 0; j < _dimension; ++j) {
			printf("%d", host_table[i][j]);
			printf(" ");
		}
		printf("\n");
	}
	printf("\n");

	// device memorian allokacio
	hipMalloc(&dev_table, dsize);
	hipMalloc(&dev_temptable, dsize);
	// memoria copy oda
	hipMemcpy(dev_table, host_table, dsize, hipMemcpyHostToDevice);

	//1d blokkok
	int blocks = (_dimension + _threads - 1) / _threads;
	//2d
	dim3 THREADS(_threads, _threads);
	dim3 BLOCKS(blocks, blocks);

	int round = 0;
	do {
		NextRound << <BLOCKS, THREADS >> > (dev_table, dev_temptable, _dimension);
		kernel << <BLOCKS, THREADS >> > (dev_table, dev_temptable, _dimension);
		++round;
	} while (round != _goal);

	// memoria copy vissza
	hipMemcpy(host_table, dev_table, dsize, hipMemcpyDeviceToHost);

	for (size_t i = 0; i < _dimension; ++i) {
		for (size_t j = 0; j < _dimension; ++j) {
			printf("%d", host_table[i][j]);
			printf(" ");
		}
		printf("\n");
	}

	printf("kesz");

	free(host_table);
	hipFree(dev_table);
	hipFree(dev_temptable);
}